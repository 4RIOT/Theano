#include "hip/hip_runtime.h"
// modified from pytorch
// https://github.com/pytorch/pytorch/master/blob/torch/lib/THC/THCTensorTopK.cuh
//
// Converts a type (maybe float) to an integer representation with the same
// sorting; i.e., for floats f1, f2:
// if f1 < f2 then convert(f1) < convert(f2)
// We use this to enable radix selection of floating-point values.
// This also gives a relative order for NaNs, but that's ok, as they
// will all be adjacent

template <typename T>
struct RadixConfig {};

template <>
struct RadixConfig<float> {
  typedef unsigned int RadixType;

  static inline __device__ RadixType convert(float v) {
    RadixType x = __float_as_int(v);
    RadixType mask = (x & 0x80000000) ? 0xffffffff : 0x80000000;

    return (x ^ mask);
  }

  static inline __device__ float deconvert(RadixType v) {
    RadixType mask = (v & 0x80000000) ? 0x80000000 : 0xffffffff;

    return __int_as_float(v ^ mask);
  }
};

template <>
struct RadixConfig<unsigned char> {
  typedef unsigned int RadixType;

  static inline __device__ RadixType convert(unsigned char v) {
    return v;
  }

  static inline __device__ unsigned char deconvert(RadixType v) {
    return v;
  }
};

template <>
struct RadixConfig<char> {
  typedef unsigned int RadixType;

  static inline __device__ RadixType convert(char v) {
    return 128u + v;
  }

  static inline __device__ char deconvert(RadixType v) {
    return v - 128;
  }
};

template <>
struct RadixConfig<short> {
  typedef unsigned int RadixType;

  static inline __device__ RadixType convert(short v) {
    assert(sizeof(short) == 2);
    return 32768u + v;
  }

  static inline __device__ short deconvert(RadixType v) {
    return v - 32768;
  }
};

template <>
struct RadixConfig<int> {
  typedef unsigned int RadixType;

  static inline __device__ RadixType convert(int v) {
    assert(sizeof(int) == 4);
    return 2147483648u + v;
  }

  static inline __device__ int deconvert(RadixType v) {
    return v - 2147483648u;
  }
};

template <>
struct RadixConfig<long> {
  typedef unsigned long long int RadixType;

  static inline __device__ RadixType convert(long v) {
    assert(sizeof(long) == 8);
    return 9223372036854775808ull + v;
  }

  static inline __device__ long deconvert(RadixType v) {
    return v - 9223372036854775808ull;
  }
};

template <>
struct RadixConfig<double> {
  typedef unsigned long long int RadixType;

  static inline __device__ RadixType convert(double v) {
    RadixType x = __double_as_longlong(v);
    RadixType mask = -((x >> 63)) | 0x8000000000000000;
    return (x ^ mask);
  }

  static inline __device__ double deconvert(RadixType v) {
    RadixType mask = ((v >> 63) - 1) | 0x8000000000000000;
    return __longlong_as_double(v ^ mask);
  }
};

#ifdef USE_HALF
template <>
struct RadixConfig<half> {
  typedef unsigned int RadixType;

  static inline __device__ RadixType convert(half v) {
#if defined(__CUDACC_VER__) && __CUDACC_VER__ >= 80000
    RadixType x = __half_as_ushort(v);
    RadixType mask = -((x >> 15)) | 0x8000;
    return (x ^ mask);
#else
    assert(false);
    return 0u;
#endif
  }

  static inline __device__ half deconvert(RadixType v) {
#if defined(__CUDACC_VER__) && __CUDACC_VER__ >= 80000
    RadixType mask = ((v >> 15) - 1) | 0x8000;
    return __ushort_as_half(v ^ mask);
#else
    assert(false);
    return ScalarConvert<int, half>::to(0);
#endif
  }
};
#endif

// $$inp_t should be replaced in c_code
// we cannot use templated __global__ because gpuarray API does not support it yet
#define NDIM            $ndim
#define INPUT_TYPE      $inp_t
#define INDEX_TYPE     $out_t
#define bitsof(T)       (sizeof(T)*8)
#define RADIX_BITS      2
#define RADIX_SIZE      (1<<RADIX_BITS)
#define RADIX_MASK(n)   ((RADIX_SIZE-1) << (n*RADIX_BITS))
#define RADIX_DIGITS(T) (bitsof(T)/RADIX_BITS)
#define radix_t         RadixConfig<INPUT_TYPE>::RadixType

#if RADIX_SIZE > 32
#error "RADIX_SIZE must be smaller than warp size (32)"
#endif

template <typename T>
static inline __device__ T binary_cumsum(int idx, int warp_id, int lane_id, T* smem, bool value) {
    // cumsum within 1D thread block, which adds up `value` of all threads whose id is *no greater than* the current thread
    // cumsum within warp
    unsigned int warp_bits = __ballot(value);
    T warp_sum = __popc(((2<<lane_id)-1) & warp_bits);

    if (lane_id == 0)
        smem[warp_id] = __popc(warp_bits);

    __syncthreads();

    // cumsum across warps in one thread
    if (idx == 0) {
        int current = 0;
        for (int i = 0; i < blockDim.x / 32; ++i) {
            T v = smem[i];
            smem[i] = smem[i]+current;
            current = current+v;
        }
    }

    __syncthreads();

    // load the carry from the preceding warp
    if (warp_id >= 1) {
        warp_sum = warp_sum+smem[warp_id - 1];
    }

    return warp_sum;
}

template <typename T>
static inline __device__ T binary_cumsum_exclusive(
    int idx, int warp_id, int lane_id, T* smem, bool value) {
    // cumsum within 1D thread block, which adds up `value` of all threads
    // whose id is *less than* the current thread

    // cumsum within warp
    unsigned int warp_bits = __ballot(value);
    T warp_sum = __popc(((1<<lane_id)-1) & warp_bits);

    if (lane_id == 0)
        smem[warp_id] = __popc(warp_bits);

    __syncthreads();

    // cumsum across warps in one thread
    if (idx == 0) {
        int current = 0;
        for (int i = 0; i < blockDim.x / 32; ++i) {
            T v = smem[i];
            smem[i] = smem[i]+current;
            current = current+v;
        }
    }

    __syncthreads();

    // load the carry from the preceding warp
    if (warp_id >= 1)
        warp_sum += smem[warp_id - 1];

    return warp_sum;
}

// apply raw(byte) offset to pointer
template <typename T>
static __device__ inline T* ptr_add(T *ptr, ga_ssize offset) {
    return (T*)((char*)ptr + offset);
}

// get array element using raw(byte) offset
template <typename T>
static __device__ inline T& ptr_at(T *ptr, ga_ssize offset) {
    return *((T*)((char*)ptr + offset));
}

KERNEL void k_topk_dense(
        $dims
        // ga_size dims_1, ga_ssize dims_2, ... , dims_$${NDIM}
        $dstv
        // INPUT_TYPE *dstv
        $dstv_strides
        // ga_ssize dstv_strides_0, ga_ssize dstv_strides_1, ... , dstv_strides_$${NDIM}
        $dsti
        // INDEX_TYPE *dsti
        $dsti_strides
        // ga_ssize dsti_strides_0, ga_ssize dsti_strides_1, ... , dsti_strides_$${NDIM}
        ga_ssize k,
        INPUT_TYPE* src,
        $src_strides
        // ga_ssize src_strides_0, ga_ssize src_strides_1, ... , src_strides_$${NDIM}
        size_t size) {
    /*
    extern __shared__ radix_t smem[];
    ga_ssize __shared__ bins[RADIX_SIZE]; // TODO: does using 32-bit gives speedup?
    bool is_topk = true;
    bool is_topkth = true; // exactly k-th largest
    radix_t out_idx;

    const size_t idx = threadIdx.x;
    size_t __shared__ k2, exceed;
    const ga_uint warp_id = idx / 32;
    const ga_uint lane_id = idx % 32;
    radix_t *wmem = (radix_t*)(smem) + warp_id * 32;
    const bool in_range = (idx < size);
    is_topk &= in_range;
    const INPUT_TYPE xval = in_range ? ptr_at(src, idx*src_strides_0) : (INPUT_TYPE)0;
    radix_t x = in_range ? RadixConfig<INPUT_TYPE>::convert(xval) : 0;

    // resolve negative k
    if (k<0) { x = ~x; k = -k; }
    if (idx==0) k2 = k;

    // 0. get the slice for thread block to work on
    size_t gid = blockIdx.x, gidx;
    $set_slice
    //for(int i=0; i<NDIM; i++) {
        //gidx = gid % dims_$${i};
        //gid /= dims_$${i};
        //dsti = ptr_add(dsti, gidx*dsti_strides_$${i+1};
        //dstv = ptr_add(dstv, gidx*dstv_strides_$${i+1};
        //src = ptr_add(src, gidx*src_strides_$${i+1});
    //}

    // 1. filter is_topk and is_topkth using radix select

    #pragma unroll
    for (int i=bitsof(INPUT_TYPE)-RADIX_BITS; i>=0; i-=RADIX_BITS) {
        smem[idx] = 0;
        int digit = (x>>i) & (RADIX_SIZE-1);
        // count within warp
        #pragma unroll
        for (int bin=0; bin<RADIX_SIZE; ++bin) {
            bool incr_bin = (bin == digit) && is_topkth && in_range;
            unsigned int incr_bin_warp = __ballot(incr_bin);
            if (lane_id==0)
                wmem[bin] += __popc(incr_bin_warp);
        }
        __syncthreads();
        // sum counts across all warps
        // TODO: test in-block parallel sum?
        if (idx < RADIX_SIZE) {
            for(int w=32; w<blockDim.x; w+=32)
                smem[idx] += smem[idx + w];
        }
        __syncthreads();

        // calculate k minus cumsum(count)
        if (idx<RADIX_SIZE)
            bins[idx] = 0;
        if (idx == 0) {
            exceed = k; // how many the number of is_topk exceeds k
            bins[RADIX_SIZE-1] = k2 - smem[RADIX_SIZE-1];
            if (bins[RADIX_SIZE-1] > 0)
                k2 = bins[RADIX_SIZE-1];
            else
                exceed = min(exceed, bins[RADIX_SIZE-1]);
            #pragma unroll
            for(int bin=RADIX_SIZE-1; bin; --bin) {
                bins[bin-1] = bins[bin] - smem[bin-1];
                if (bins[bin-1] > 0)
                    k2 = bins[bin-1];
                else
                    exceed = min(exceed, bins[bin-1]);
            }
        }
        __syncthreads();


        // smem -> count
        // bins -> k2 - cumsum(count)
        if (is_topk && is_topkth) {
            ga_ssize icount = bins[digit];
            if (icount > 0) {
                is_topkth = false;
            } else if (icount < 0) {
                if (digit+1!=RADIX_SIZE) {
                    if (bins[digit+1] <= 0) {
                        is_topk = false;
                        is_topkth = false;
                    }
                }
            }
        }
    }

    // 2. find the index of output array, if exists

    if (exceed != 0) {
        // top_kth value may not be unique, so we need to
        // perform binary cumsum on is_topkth to drop exceeding top-kth values
        out_idx = binary_cumsum_exclusive<radix_t>(idx, warp_id, lane_id, smem, is_topkth);
        is_topk &= (out_idx < exceed);
    }

    // perform binary cumsum on is_topk to determine the indices to put result
    out_idx = binary_cumsum_exclusive<radix_t>(idx, warp_id, lane_id, smem, is_topk);
    __syncthreads();

    if (is_topk) {
        $write_value;
        // ptr_at(dstv, out_idx * dstv_strides_0) = xval;
        $write_index;
        // ptr_at(dsti, out_idx * dsti_strides_0) = (INDEX_TYPE)idx;
    }
    */
}
